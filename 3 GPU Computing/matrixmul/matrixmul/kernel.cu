#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>

#define A 2
#define B 3
#define C 4

__global__ void product(float *a, float *b, float *c, int aa, int bb, int cc)
{
	int ix = threadIdx.x + blockIdx.x * blockDim.x;
	int iy = threadIdx.y + blockIdx.y * blockDim.y;
	if (ix < aa && iy < cc)
	{
		float sum = 0;
		for (int index = 0; index < bb; index++)
		{
			sum += a[ix * bb + index] * b[index * cc + iy];
		}
		c[ix * cc + iy] = sum;
	}
}

int main()
{
	//********** CPU, matrix initialization **********
	float *a, *b, *c;

	a = (float *)malloc(A * B*sizeof(float));
	b = (float *)malloc(B * C*sizeof(float));
	c = (float *)malloc(A * C*sizeof(float));

	for (int i = 0; i < A * B; ++i)
		a[i] = 2;
	for (int i = 0; i < B * C; ++i)
		b[i] = 2;
	for (int i = 0; i < A * C; ++i)
		c[i] = 0;
	
	printf("input matrix 1:\n");
	for (int i = 0; i < A; i++)
	{
		for (int j = 0; j < B; j++)
			printf("%.2f\t", a[i * B + j]);
		printf("\n");
	}
	printf("\n");
	printf("input matrix 2:\n");
	for (int i = 0; i < B; i++)
	{
		for (int j = 0; j < C; j++)
			printf("%.2f\t", b[i * C + j]);
		printf("\n");
	}
	
	clock_t startc, finishc;

	//start using CPU to multiply matrix
	startc = clock();
	for (int i = 0; i < A; i++){
		for (int j = 0; j < C; j++)
		{
			float sum = 0;
			for (int k = 0; k < B; k++)
			{
				sum += a[i * B + k] * b[k * C + j];
			}
			c[i * C + j] = sum;
		}
	}
	finishc = clock();

	printf("\n");
	printf("output matrix (CPU):\n");
	//printf("number in matrix: %.2f\n", c[10]);
	
	for (int i = 0; i < A; i++)
	{
		for (int j = 0; j < C; j++)
			printf("%.2f\t", c[i * B + j]);
		printf("\n");
	}

	printf("***********************************************\n");
	printf("The total time using CPU: %f seconds\n", ((float)finishc - startc) / 1000);
	printf("***********************************************\n");

	//********** GPU, matrx initialization **********
	clock_t start, finish;
	float *d_a, *d_b, *d_c;

	//start using GPU to multiply matrix
	start = clock();

	hipMalloc(&d_a, A * B*sizeof(float));
	hipMalloc(&d_b, B * C*sizeof(float));
	hipMalloc(&d_c, A * C*sizeof(float));

	hipMemcpy(d_a, a, A * B*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, B * C*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, A * C*sizeof(float), hipMemcpyHostToDevice);

	int dimx = 32;
	int dimy = 32;
	dim3 block(dimx, dimy);
	dim3 grid((A + block.x - 1) / block.x, (C + block.y - 1) / block.y);

	product << <grid, block >> >(d_a, d_b, d_c, A, B, C);

	hipMemcpy(c, d_c, A * C*sizeof(float), hipMemcpyDeviceToHost);
	/*
	printf("\n");
	printf("output matrix (using GPU):\n");
	for (int i = 0; i < A; i++)
	{
		for (int j = 0; j < C; j++)
			printf("%.2f\t", c[i * C + j]);
		printf("\n");
	}
	*/
	free(a);
	free(b);
	free(c);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	finish = clock();
	//printf("number in matrix (GPU): %d\n", f[1]);
	printf("***********************************************\n");
	printf("The total time using GPU: %f seconds\n", ((float)finish - start) / 1000);
	printf("***********************************************\n");

	return 0;
}