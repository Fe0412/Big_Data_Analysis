#include "hip/hip_runtime.h"

#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define A 6
#define B 2
#define C 1

__global__ void transpose(float *odata, float* idata, int ny, int nx)
{
	int ix = blockDim.x * blockIdx.x + threadIdx.x;
	int iy = blockDim.y * blockIdx.y + threadIdx.y;

	if (ix < nx && iy < ny)
	{
		odata[ix * ny + iy] = idata[iy * nx + ix];
	}
}

__global__ void product(float *a, float *b, float *c, int aa, int bb, int cc)
{
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int iy = blockIdx.y * blockDim.y + threadIdx.y;
	if (ix < aa && iy < cc)
	{
		float sum = 0;
		for (int index = 0; index < bb; index++)
		{
			sum += a[ix * bb + index] * b[index * cc + iy];
		}
		c[ix * cc + iy] = sum;
	}
}

#define PERR(call) \
if (call) {\
	fprintf(stderr, "%s:%d Error [%s] on "#call"\n", __FILE__, __LINE__, \
	hipGetErrorString(hipGetLastError())); \
	exit(1); \
}
#define ERRCHECK \
if (hipPeekAtLastError()) {\
	fprintf(stderr, "%s:%d Error [%s]\n", __FILE__, __LINE__, \
	hipGetErrorString(hipGetLastError())); \
	exit(1); \
}

__global__ void inv_kernel(float *a_i, float *c_o, int n)
{
	int *p = (int *)malloc(3 * sizeof(int));
	int *info = (int *)malloc(sizeof(int));
	int batch;
	hipblasHandle_t hdl;
	hipblasStatus_t status = hipblasCreate(&hdl);

	info[0] = 0;
	batch = 1;
	float **a = (float **)malloc(sizeof(float *));
	*a = a_i;
	const float **aconst = (const float **)a;
	float **c = (float **)malloc(sizeof(float *));
	*c = c_o;
	status = hipblasSgetrfBatched(hdl, n, a, n, p, info, batch);
	__syncthreads();
	status = hipblasSgetriBatched(hdl, n, aconst, n, p,
		c, n, info, batch);
	__syncthreads();
	hipblasDestroy(hdl);
}

static void run_inv(float *in, float *out, int n)
{
	float *a_d, *c_d;

	PERR(hipMalloc(&a_d, n*n*sizeof(float)));
	PERR(hipMalloc(&c_d, n*n*sizeof(float)));
	PERR(hipMemcpy(a_d, in, n*n*sizeof(float), hipMemcpyHostToDevice));

	inv_kernel << <1, 1 >> >(a_d, c_d, n);

	hipDeviceSynchronize();
	ERRCHECK;

	PERR(hipMemcpy(out, c_d, n*n*sizeof(float), hipMemcpyDeviceToHost));
	PERR(hipFree(a_d));
	PERR(hipFree(c_d));
}


int main(int argc, char **argv)
{
	//-------------------- matrix transpose X' --------------------
	float X[12] = { 1, 0, 1, 1, 1, 2, 1, 3, 1, 4, 1, 5 };
	printf("X:\n");
	for (int i = 0; i < 6; i++)
	{
		for (int j = 0; j < 2; j++)
			printf("%.4f\t", X[i * 2 + j]);
		printf("\n");
	}
	float y[6] = { 0, 20, 60, 68, 77, 110 };
	printf("\n");
	printf("y:\n");
	for (int i = 0; i < 6; i++)
	{
		printf("%.4f\t", y[i]);
	}
	printf("\n");
	printf("\n");

	float *b;
	//b is transpose of x
	b = (float *)malloc(B * A*sizeof(float));

	for (int i = 0; i < B * A; ++i)
		b[i] = 0;

	//-------------------- matrx initialization for transpose --------------------
	float *d_a, *d_b;

	hipMalloc(&d_a, A * B*sizeof(float));
	hipMalloc(&d_b, B * A*sizeof(float));

	hipMemcpy(d_a, X, A * B*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, B * A*sizeof(float), hipMemcpyHostToDevice);

	int dimx = 32;
	int dimy = 32;
	dim3 block(dimx, dimy);
	dim3 grid((B + block.x - 1) / block.x, (A + block.y - 1) / block.y);

	transpose << <grid, block >> >(d_b, d_a, A, B);

	hipMemcpy(b, d_b, B * A*sizeof(float), hipMemcpyDeviceToHost);

	printf("X':\n");
	for (int i = 0; i < B; i++)
	{
		for (int j = 0; j < A; j++)
			printf("%.4f\t", b[i * A + j]);
		printf("\n");
	}

	//-------------------- matrx initialization for multiplication X'*X--------------------
	float *c;
	c = (float *)malloc(B * B*sizeof(float));	
	float *d_c;

	//start using GPU to multiply matrix
	hipMalloc(&d_c, B * B*sizeof(float));
	hipMemcpy(d_c, c, B * B*sizeof(float), hipMemcpyHostToDevice);

	dim3 grid2((B + block.x - 1) / block.x, (B + block.y - 1) / block.y);

	product << <grid2, block >> >(d_b, d_a, d_c, B, A, B);

	hipMemcpy(c, d_c, B * B*sizeof(float), hipMemcpyDeviceToHost);

	printf("\n");
	printf("X'*X:\n");
	for (int i = 0; i < B; i++)
	{
		for (int j = 0; j < B; j++)
			printf("%.4f\t", c[i * B + j]);
		printf("\n");
	}

	//-------------------- pinv(X'*X) --------------------
	float *invmatrix;
	invmatrix = (float *)malloc(B * B*sizeof(float));
	run_inv(c, invmatrix, B);
	printf("\n");
	printf("pinv(X'*X):\n");
	for (int i = 0; i < B; i++){
		for (int j = 0; j < B; j++) printf("%.4f, ", invmatrix[(B * i) + j]);
		printf("\n");
	}

	//-------------------- pinv(X'*X)*X' --------------------
	float *invma;
	hipMalloc(&invma, B * B*sizeof(float));
	hipMemcpy(invma, invmatrix, B * B*sizeof(float), hipMemcpyHostToDevice);

	float *e;
	e = (float *)malloc(B * A*sizeof(float));
	float *d_e;
	hipMalloc(&d_e, B * A*sizeof(float));
	hipMemcpy(d_e, e, B * A*sizeof(float), hipMemcpyHostToDevice);

	dim3 grid3((B + block.x - 1) / block.x, (A + block.y - 1) / block.y);
	product << <grid3, block >> >(invma, d_b, d_e, B, B, A);
	hipMemcpy(e, d_e, B * A*sizeof(float), hipMemcpyDeviceToHost);

	printf("\n");
	printf("pinv(X'*X)*X:\n");
	for (int i = 0; i < B; i++)
	{
		for (int j = 0; j < A; j++)
			printf("%.4f\t", e[i * A + j]);
		printf("\n");
	}

	//-------------------- pinv(X'*X)*X'*y' --------------------
	float *res;
	res = (float *)malloc(B * C*sizeof(float));
	float *d_res;
	hipMalloc(&d_res, B * C*sizeof(float));
	hipMemcpy(d_res, res, B * C*sizeof(float), hipMemcpyHostToDevice);

	float *d_y;
	hipMalloc(&d_y, A * C*sizeof(float));
	hipMemcpy(d_y, y, A * C*sizeof(float), hipMemcpyHostToDevice);

	dim3 grid4((B + block.x - 1) / block.x, (C + block.y - 1) / block.y);
	product << <grid4, block >> >(d_e, d_y, d_res, B, A, C);
	hipMemcpy(res, d_res, B * C*sizeof(float), hipMemcpyDeviceToHost);

	printf("\n");
	printf("the result is:\n");
	for (int i = 0; i < B; i++)
	{
		for (int j = 0; j < C; j++)
			printf("%.4f\t", res[i * C + j]);
		printf("\n");
	}

	printf("\n");
	printf("------------------------------------------\n");
	printf("The regression model is:");
	printf("y = %.4f * x + %.4f\n", res[1],res[0]);
	printf("------------------------------------------\n");

	free(b);
	free(c);
	free(invmatrix);
	free(e);
	free(res);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	hipFree(invma);
	hipFree(d_e);
	hipFree(d_res);

	return 0;
}